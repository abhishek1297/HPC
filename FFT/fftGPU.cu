#include <cstdio>
#include <iostream>
#include <algorithm>
#include <hipfft/hipfft.h>
#include <hip/hip_runtime.h>

#include <cuda_device_runtime_api.h>

#define N 16
typedef float2 Complex;

int main () {

    const int MEM_SIZE = sizeof(Complex) * N;
    //----------------Initialize-----------------
    Complex *h_data = reinterpret_cast<Complex *>(malloc(MEM_SIZE)); //float2
    std::fill(h_data, h_data + N, make_float2(0.0, 0.0));
    for (int i = 0; i < 5; ++i) {
        h_data[i].x = h_data[N - i].x = 1.0;
    }

    printf("Original signal:\n");
    for (int i = 0; i < N; ++i) {
        printf("%d %e %e\n", i, h_data[i].x, h_data[i].y);
    }

    Complex *d_data;
    hipMalloc(reinterpret_cast<void **>(&d_data), MEM_SIZE);
    hipMemcpy(d_data, h_data, MEM_SIZE, hipMemcpyHostToDevice);

    hipfftHandle plan;
    hipfftPlan1d(&plan, N, HIPFFT_C2C, 1);
    //----------------Perform FFT----------------------
    hipfftExecC2C(plan, reinterpret_cast<hipfftComplex *>(d_data),
    reinterpret_cast<hipfftComplex *>(d_data),
    HIPFFT_FORWARD);
    hipMemcpy(h_data, d_data, MEM_SIZE, hipMemcpyDeviceToHost);
    printf("\n\nAfter GPU FFT:\n");
    for (int i = 0; i < N; ++i) {
        printf("%d %e %e\n", i, h_data[i].x/sqrt(N), h_data[i].y/sqrt(N));
    }
    //----------------Perform Inverse FFT--------------
    hipfftExecC2C(plan, reinterpret_cast<hipfftComplex *>(d_data),
    reinterpret_cast<hipfftComplex *>(d_data),
    HIPFFT_BACKWARD);

    hipMemcpy(h_data, d_data, MEM_SIZE, hipMemcpyDeviceToHost);
    printf("\n\nAfter GPU Inverse FFT:\n");
    for (int i = 0; i < N; ++i) {
        printf("%d %e %e\n", i, h_data[i].x, h_data[i].y);
    }
    hipFree(d_data);
    free(h_data);

    return 0;
}